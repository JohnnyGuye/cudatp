#include "hip/hip_runtime.h"
#include "wb.h"

#include <fstream>
#include <cmath>

#define SQR(x) ((x)*(x))

class Vector3 {
public:
	float r;
	float g;
	float b;

	__host__ __device__ Vector3( float r, float g, float b )
	: r(r), g(g), b(b) {}
	__host__ __device__ Vector3( float rgb = 0 )
	: r(rgb), g(rgb), b(rgb) {}
	
	__host__ __device__ static float distance2( const Vector3 & lhs, const Vector3 &rhs ) {
		return SQR(lhs.r - rhs.r) + SQR(lhs.g - rhs.g) + SQR(lhs.b - rhs.b);		
	}

	__host__ __device__ static float dotProduct( const Vector3 & lhs, const Vector3 & rhs ) {
		return lhs.r * rhs.r + lhs.g * rhs.g + lhs.b * rhs.b; 
	}

	__host__ __device__ static Vector3 normalized( const Vector3 & v ) {
		auto sqr = sqrt( Vector3::dotProduct( v, v ) );
		return Vector3( v.r / sqr, v.g / sqr, v.b / sqr );
	}

};

class Sphere {
public:
	float size = 1;
	Vector3 center;
	Vector3 color = Vector3( 1.0, 0.3, 0.4 );
};


__global__ void raytrace( Vector3 * image, Sphere * sphere, int width, int height, int sphereCount ) {

	int i = threadIdx.x;
	int j = blockIdx.x;

	if( i > width ) return;
	if( j > height ) return;
	
	Vector3 pos = Vector3( i, j, 0 );
	auto dist2 = Vector3::distance2( pos, sphere.center );
	auto sqrSize2 = SQR( sphere.size );
	if( dist2 > sqrSize2 ){
		
		image[ j * width + i ] = Vector3( 1 - (j / (float)height) );		

	} else {

		image[ j * width + i ] = sphere.color;
		image[ j * width + i ].r *= 1 - dist2 / sqrSize2;
		image[ j * width + i ].g *= 1 - dist2 / sqrSize2;
		image[ j * width + i ].b *= 1 - dist2 / sqrSize2;
	
	}
	
}

void writeP3( std::string fileName, Vector3 * data, int width, int height ) {
	
	std::ofstream file;
	file.open( fileName );
	file << "P3\n" << width << " " << height << "\n" << 255 << std::endl;
	
	for( auto idx = 0; idx < width * height; idx++ ) {
		file << (int)(data[ idx ].r * 255) << " "
			<< (int)(data[ idx ].g * 255) << " "
			<< (int)(data[ idx ].b * 255) << " ";
	}

	file << std::flush;

}

int main(int argc, char *argv[]) {

	Vector3 * imgData;
	Vector3 * d_imgData;

	int width = 256;
	int height = 256;

	Sphere * sphere = new Sphere[1];

	Sphere sphere;
	sphere.center.r = 128;
	sphere.center.g = 128;
	sphere.size		= 60;
	
	imgData = new Vector3[ width * height ];
	
	hipMalloc( (void **) &d_imgData, width * height * sizeof( Vector3 ) );	

	raytrace<<< 256, 256 >>>( d_imgData, sphere, width, height );

	hipMemcpy( imgData, d_imgData, width * height * sizeof( Vector3 ), hipMemcpyDeviceToHost );	

	writeP3( "output.ppm", imgData, width, height );

	delete [] imgData;

}
