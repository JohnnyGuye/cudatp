#include "hip/hip_runtime.h"
#include "wb.h"

#define SQR(x) ((x)*(x))
#define SQR5(x) ((x)*(x)*(x)*(x)*(x))

#include "cuda.def.cuh"
#include "vector3.h"
#include <fstream>
#include <cmath>


// ==========================================================
// ============ COMPILATION OPTIONS =========================
// ==========================================================

const bool OPT_CPU_MAXLUMINANCE = false;

// ==========================================================
// ============ CLASSES DEFINITIONS =========================
// ==========================================================

class Ray {
public:
    Vector3 origin;
    Vector3 direction;

    GCPU_F Ray( const Vector3 & origin, const Vector3 & direction )
        : origin( origin ), direction( direction ) {}

    GCPU_F Ray( const Vector3 & direction )
        : Ray( 0, direction ) {}

    GCPU_F Ray( )
        : Ray( Vector3(1, 0, 0) ) {}

};

class Material {
public:
    Vector3 albedo;
    float roughness = 0;
    float metalness = 0;
};

class Camera {
public:
    Vector3 resolution  = Vector3(100,100,1);
    Vector3 direction   = Vector3(0,0,1);
    Vector3 up          = Vector3(0,1,0);
    Vector3 position    = Vector3(0,0,-100);

    bool orthogonal = true;
    Vector3 size = Vector3(100,100,0);

    GCPU_F Ray castRayFromPixel( int x, int y ) {
        if( orthogonal ) {
            Ray ray;
            ray.origin = (Vector3(x,y,0) / resolution - 0.5f) * size + position;
            ray.direction = direction;
            return ray;
        } else {
            Ray ray;
            return ray;
        }
    }

};

class Sphere {
public:
        float size = 1;
	Vector3 center;
        Material material;

        GCPU_F Sphere()
            : size(1), center( 0.0 ){}

        GCPU_F Sphere( Sphere & sphere )
            : size( sphere.size ), center( sphere.center ), material( sphere.material ) {}

        GCPU_F Vector3 normalAt( const Vector3 & hit ) const {
            return (hit - center).normalize();
        }

        GCPU_F bool intersects( const Vector3 & origin, const Vector3 & direction, Vector3 & hit, float & distance ) const {
            auto m = origin - center;
            float b = Vector3::dot( m, direction );
            float c = m.length2() - SQR( size );

            if( c > 0.0f && b > 0.0f ) return false;
            float discr = SQR( b ) - c;

            if( discr < 0.0f ) return false;
            auto t = -b - sqrt( discr );

            if( t < 0.0f ) t = -b + sqrt( discr );

            hit = origin + direction * t;
            distance = t;

            return true;
        }

        friend std::ostream & operator << ( std::ostream & os, const Sphere & sphere ) {
            os << "Sphere: radius: " << sphere.size
               << " center: " << sphere.center
               << " color: " << sphere.material.albedo;
            return os;
        }
};

class Light {
public:
    Vector3 origin;
    Vector3 color = 1;
    float intensity = 1;

    Light() {}
};

class Collision {
public:

    Sphere * sphere = nullptr;
    Vector3 normal;
    Vector3 hit;
    float distance = 2 * 1e38;

};

GCPU_F bool getFirstCollision( const Ray & ray, Sphere * spheres, const int sphereCount, Collision & collision ) {

    collision.distance = 2e38;
    collision.sphere = nullptr;

    auto collided = false;

    for( unsigned int i = 0; i < sphereCount; i++ ) {
        Vector3 hit;
        float distance;
        if( !spheres[i].intersects( ray.origin, ray.direction, hit, distance ) ) continue;
        if( collision.distance < distance ) continue;

        collision.distance = distance;
        collision.hit = hit;
        collision.normal = spheres[i].normalAt( hit );
        collision.sphere = &spheres[i];
        collided = true;

    }

    return collided;
}

GCPU_F float GGXDistribution( float alpha2, float NdotH ) {
    auto denom = SQR( NdotH ) * (alpha2 - 1) + 1;
    return alpha2 / (M_PI * SQR( denom ));
}

GCPU_F float GGXGeometry( float alpha2, float NdotL ) {
    return 2 * NdotL / (NdotL + sqrt(alpha2 + (1-alpha2) * SQR(NdotL) ));
}

GCPU_F Vector3 Fresnel( const Vector3 & f0, const float NdotH ) {
    auto unary = Vector3(1);
    return f0 + (unary - f0) * (SQR5(unary - NdotH));
}

GCPU_F float dodgeZero( float value ) {
    if( value  < 0 ) value = 0;
    if( value < 0.001 ) value = 0.001;
    return value;
}

GCPU_F float computeLuminance( const Vector3 & color ) {
    return color.dot(Vector3(0.2126f, 0.7152f, 0.0722f));
}

__global__ void raytrace( Vector3 * image, Sphere * spheres, Light * lights, Camera camera, int sphereCount, int lightCount ) {

        int width = camera.resolution.x;
        int height = camera.resolution.y;

        int idx = threadIdx.x
                + threadIdx.y * blockDim.y
                + blockIdx.x * blockDim.x * blockDim.y;

        if( idx >= width * height ) return;
        
	//printf("%d\n", threadIdx.x + threadIdx.y * blockDim.x);
        

        int i = idx % width;
        int j = height - idx / height;

//        printf("%d %d %d\n", i, j, idx );
	
        //printf("-- %d %d %d\n", spheres[0].center.r, spheres[0].center.g, spheres[0].center.b );

        Ray ray;
        ray.origin = Vector3( i, j, -100 );
        ray.direction = Vector3( 0, 0, 1 );

        ray = camera.castRayFromPixel( i, j );

//        printf("%f %f %f\n", ray.origin.x, ray.origin.y, ray.origin.z);
//        image[ idx ] = (ray.origin / camera.resolution * 2 + 1).abs();
        image[ idx ] = ((ray.origin - camera.position) / camera.size).abs();

        Collision collision;
        if( !getFirstCollision( ray, spheres, sphereCount, collision ) ) return;

        //image[ idx ] = collision.sphere->color;

        auto material = collision.sphere->material;
        auto albedo = material.albedo;
        auto rough = material.roughness * 0.97 + 0.03;
        auto metal = material.metalness;
        auto f0 = albedo * (0.04 * (1-metal) + metal);

        auto alpha = SQR( rough );
        auto alpha2 = SQR( alpha );

        image[ idx ] = 0;

        for( unsigned int i = 0; i < lightCount; i++ ) {
            auto & light = lights[i];
            auto color = light.color * light.intensity;
            auto L = (light.origin - collision.hit).normalize();
            auto N = collision.normal;
            auto V = -ray.direction.normalized();
            auto H = (L + V).normalize();

            auto NdotL = Vector3::dot( N, L );

            if( NdotL <= 0 ) continue;

            auto NdotV = dodgeZero( N.dot(V) );
            auto NdotH = dodgeZero( N.dot(H) );

            NdotL = dodgeZero( NdotL );

            auto D = GGXDistribution( alpha2, NdotL );
            auto G = GGXGeometry( alpha2, NdotL ) * GGXGeometry( alpha2, NdotV );
            auto F = Fresnel( f0, NdotH );
            auto denom = 4 * NdotL * NdotV;

            auto spec = F * G * D / denom;
            auto nonspec = (Vector3(1) - F);

            color *= albedo * (spec + nonspec) / M_PI * NdotL;

            image[idx] += color;
        }	
}

__global__ void tonemap( Vector3 * image, float maxLuminance, Camera camera ) {

    int width = camera.resolution.x;
    int height = camera.resolution.y;

    int idx = threadIdx.x
            + threadIdx.y * blockDim.y
            + blockIdx.x * blockDim.x * blockDim.y;

    if( idx >= width * height ) return;

//    int i = idx % width;
//    int j = height - idx / height;

    auto coeff = (1 + maxLuminance) / maxLuminance;
    auto luminance = computeLuminance(image[idx]);
    image[idx] = image[idx] * luminance / (1 + luminance) * coeff;

}

// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
template< int VERS >
__global__ void computeMaxLuminance( const Vector3 * in_colors, Vector3 * out_colors, const Camera camera ) {

    extern __shared__ Vector3 sharedData[];

    unsigned int tid    = threadIdx.x;
    unsigned int i      = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int px     = camera.resolution.x * camera.resolution.y;
    if( i >= px || tid >= px ) return;

    sharedData[ tid ] = in_colors[ i ];
    __syncthreads();

    auto lumiL = computeLuminance( sharedData[tid] );

    // The reduction
    for( unsigned int s = 1; s < blockDim.x; s *= 2 ) {
        if( tid % (2*s) == 0) {
            auto lumiR = computeLuminance( sharedData[tid+s] );
            if( lumiR > lumiL )
                sharedData[ tid ] = sharedData[tid+s];
        }
        __syncthreads();
    }

    if( tid == 0 ) out_colors[blockIdx.x] = sharedData[0];
}

void writeP3( std::string fileName, Vector3 * data, int width, int height ) {
	
	std::ofstream file;
	file.open( fileName );
	file << "P3\n" << width << " " << height << "\n" << 255 << std::endl;
	
        for( auto idx = 0; idx < width * height; idx++ ) {
            auto d = data[ idx ];
            d.clamp( 0, 1 );
//            std::cout << d * 255 << std::endl;
            file << (int)(d.r * 255) << " "
                    << (int)(d.g * 255) << " "
                    << (int)(d.b * 255) << " ";
	}

	file << std::flush;

}

class Scene {
public:

    Light * lights;
    int lightCount = 0;

    Sphere * spheres;
    int sphereCount = 0;

    GCPU_F Scene() {}

    GCPU_F ~Scene() {
        if( sphereCount != 0 ) delete [] spheres;
        if( lightCount != 0 ) delete [] lights;
    }

    Scene & addSphere( const Sphere & sphere ) {
        Sphere * spheres = new Sphere[ sphereCount + 1 ];
        for( unsigned int i = 0; i < sphereCount; i++ ) {
            spheres[i] = this->spheres[i];
        }

        spheres[sphereCount] = sphere;
        if( sphereCount ) {
            delete [] this->spheres;
        }

        this->spheres = spheres;
        sphereCount++;
        return *this;
    }

    Scene & addLight( const Light & light ) {
        Light * lights = new Light[ lightCount + 1 ];
        for( unsigned int i = 0; i < lightCount; i++ ) {
            lights[i] = this->lights[i];
        }

        lights[lightCount] = light;
        if( lightCount ) {
            delete [] this->lights;
        }

        this->lights = lights;
        lightCount++;
        return *this;
    }

};

void populateScene( Scene & scene ) {

    auto steps = 5;
    auto stepW = 300 / (steps + 2);
    auto stepH = 300 / (steps + 2);
    for( unsigned int i = 0; i <= steps; i++ ) {
        for( unsigned int j = 0; j <= steps; j++ ) {

            Sphere sphere;
            sphere.center   = Vector3( stepW * (i + 1), stepH * (j + 1), 0 );
            sphere.size     = min( stepW, stepH ) / 2.5f;

            sphere.material.albedo = Vector3(1, 0.6, 0.8);
            sphere.material.roughness = (float)i/(float)steps;
            sphere.material.metalness = (float)j/(float)steps;

            scene.addSphere( sphere );
        }
    }
//    Sphere sphere1;
//    sphere1.center = Vector3( 140, 130, 0 );
//    sphere1.size = 100;
//    sphere1.material.albedo = Vector3( 1, 0, 0 );

//    Sphere sphere2;
//    sphere2.center = Vector3( 210, 230, 0 );
//    sphere2.size = 20;
//    sphere2.material.albedo = Vector3( 0, 0, 1 );

//    scene.addSphere( sphere1 );

//    scene.addSphere( sphere2 );

    Light light1;
    light1.origin = Vector3( 0, 10000, -10000 );
    light1.color = Vector3( 0.6, 0.8, 1 );
    light1.intensity = 5;

    Light light2;
    light2.origin = Vector3( 500, 500, -500 );
    light2.color = Vector3( 1, 0.8, 0.6 );
    light2.intensity = 5;

    scene.addLight( light1 );
    scene.addLight( light2 );

}


int main(int argc, char *argv[]) {

        hipDeviceReset();

	Vector3 * imgData;

	Vector3 * d_imgData;
        Sphere * d_spheres;
        Light * d_lights;

        Camera camera;
        Scene scene;
        populateScene( scene );

        camera.resolution = Vector3( 1024, 1024, 0) / 16  + Vector3(0,0,1);
        camera.size = Vector3( 300, 300, 1 );
        camera.position = Vector3( 150, 150, -100 );

        int width = camera.resolution.x;
        int height = camera.resolution.y;
        auto pixelCount = width * height;

        imgData = new Vector3[ pixelCount ];
	
        hipMalloc( (void **) &d_imgData, pixelCount * sizeof( Vector3 ) );
        hipMalloc( (void **) &d_spheres, scene.sphereCount * sizeof( Sphere ) );
        hipMalloc( (void **) &d_lights, scene.lightCount * sizeof( Light ) );

        hipMemcpy( d_spheres, scene.spheres, scene.sphereCount * sizeof( Sphere ), hipMemcpyHostToDevice );
        hipMemcpy( d_lights, scene.lights, scene.lightCount * sizeof( Light ), hipMemcpyHostToDevice );

        dim3 threaddim = dim3( 32, 32, 1 );
        auto totalBlockCount = ceil( pixelCount / (float)(threaddim.x * threaddim.y) );
        dim3 blockdim = dim3( totalBlockCount, 1, 1 );

        raytrace<<< blockdim, threaddim >>>( d_imgData, d_spheres, d_lights, camera, scene.sphereCount, scene.lightCount );

        float maxLuminance = 0;

        if( OPT_CPU_MAXLUMINANCE ) {
            // CPU max
            hipDeviceSynchronize();
            hipMemcpy( imgData, d_imgData, pixelCount * sizeof( Vector3 ), hipMemcpyDeviceToHost );
            for( unsigned int i = 0; i < pixelCount; i++ ) {
                auto luminance = computeLuminance( imgData[i] );
                if( luminance > maxLuminance ) maxLuminance = luminance;
            }
            hipMemcpy( d_imgData, imgData, pixelCount * sizeof( Vector3 ), hipMemcpyHostToDevice );

        } else {
            // GPU max
            dim3 threaddim = dim3( 256, 1, 1 );
            dim3 blockdim = dim3( ceil( pixelCount / (float)(threaddim.x) ), 1, 1 );

            Vector3 * d_reduction;
            Vector3 * d_reduction2;
            Vector3 * reduction = new Vector3[ pixelCount ];

            hipDeviceSynchronize();
            hipMalloc( (void **) &d_reduction, pixelCount * sizeof( Vector3 ) );
//            hipMalloc( (void **) &d_reduction2, pixelCount * sizeof( Vector3 ) );
            CHECK_ERROR

            computeMaxLuminance<1><<< blockdim, threaddim, pixelCount * sizeof( Vector3 ) >>>( d_imgData, d_reduction, camera );

            CHECK_ERROR
            hipMemcpy( imgData, d_imgData, pixelCount * sizeof( Vector3 ), hipMemcpyDeviceToHost );
            CHECK_ERROR
            hipMemcpy( reduction, d_reduction, pixelCount * sizeof( Vector3 ), hipMemcpyDeviceToHost );
            CHECK_ERROR

            for( unsigned int i = 0; i < pixelCount; i += 2 ) {
                if( imgData[i] == reduction[i] ) continue;
                std::cout << "> " << i << std::endl;
                std::cout << imgData[i] << " " << imgData[i+1] << std::endl;
                std::cout << reduction[i] << " " << reduction[i+1] << std::endl;
            }
            hipFree( d_reduction );
            CHECK_ERROR

            maxLuminance = 1;
        }

        CHECK_ERROR

        tonemap<<< blockdim, threaddim >>>( d_imgData, maxLuminance, camera );

        hipDeviceSynchronize();
        hipMemcpy( imgData, d_imgData, pixelCount * sizeof( Vector3 ), hipMemcpyDeviceToHost );

        CHECK_ERROR

        hipFree( d_imgData );
        hipFree( d_spheres );
        hipFree( d_lights );

        writeP3( "output.ppm", imgData, width, height );

        delete [] imgData;
}
